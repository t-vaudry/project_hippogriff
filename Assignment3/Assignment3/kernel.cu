#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>
#include "Dependencies\glew\glew.h"
#include "Dependencies\freeglut\freeglut.h"
#include <stdlib.h>
#include <random>
#include <stdio.h>

using namespace std;

#define WIDTH 1024
#define HEIGHT 768
#define SIZE 786432
#define X_SIZE 0.001953125 // 2/1024
#define Y_SIZE 0.002604166 // 2/768

int numOfSpecies;
float** color = new float*[10];
bool* species;
bool* tmp;
bool* dev_species = 0;
bool* dev_tmp = 0;

void glutTimer(int value); 
void initializeColor();
void initializeGrid();
void keyboard(unsigned char key, int x, int y);
void display();
void draw();
void gameOfLife();

__global__ void countNeighbors(bool* dev_s, bool* temp, int type)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
	int numOfNeighbors = 0;

	if (i >= WIDTH)
	{
		if (i%WIDTH != 0)
		{
			if (dev_s[type*SIZE + i - WIDTH - 1]) // Top-Left Corner
				numOfNeighbors++;
		}
		if (dev_s[type*SIZE + i - WIDTH]) // Top-Center Edge
			numOfNeighbors++;
		if (i%WIDTH != WIDTH - 1)
		{
			if (dev_s[type*SIZE + i - WIDTH + 1]) // Top-Right Corner
				numOfNeighbors++;
		}
	}

	if (i < WIDTH*(HEIGHT - 1))
	{
		if (i%WIDTH != 0)
		{
			if (dev_s[type*SIZE + i + WIDTH - 1]) // Bottom-Left Corner
				numOfNeighbors++;
		}
		if (dev_s[type*SIZE + i + WIDTH]) // Bottom-Center Edge
			numOfNeighbors++;
		if (i%WIDTH != WIDTH - 1)
		{
			if (dev_s[type*SIZE + i + WIDTH + 1]) // Bottom-Right Corner
				numOfNeighbors++;
		}
	}

	if (i%WIDTH != 0)
	{
		if (dev_s[type*SIZE + i - 1]) // Middle-Left Edge
			numOfNeighbors++;
	}

	if (i%WIDTH != WIDTH - 1)
	{
		if (dev_s[type*SIZE + i + 1]) // Middle-Right Edge
			numOfNeighbors++;
	}

	// Rules of game of life for next state
	if (numOfNeighbors < 2) // Less than 2, underpopulated
		temp[i + type*SIZE] = false;
	else if ((numOfNeighbors == 2) && (!dev_s[type*SIZE + i])) // 2 neighbors, and currently dead, remain dead
		temp[i + type*SIZE] = false;
	else if (numOfNeighbors == 3) // 3 neighbors, revive/remain alive
		temp[i + type*SIZE] = true;
	else if (numOfNeighbors > 3) // More than 3, overpopulated
		temp[i + type*SIZE] = false;
}

__global__ void setState(bool* dev_s, bool* temp)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	dev_s[i] = temp[i];
}

int main(int argc, char** argv)
{
	cout << "Enter number of species: " << endl;
	cin >> numOfSpecies;

	if (numOfSpecies < 5)
	{
		cout << "Number of species is less than 5. Default to 5." << endl;
		numOfSpecies = 5;
	}
	else if (numOfSpecies > 10)
	{
		cout << "Number of species is greater than 10. Default to 10." << endl;
		numOfSpecies = 10;
	}

	glutInit(&argc, argv);
	// Initialize the color schemes and initial grid
	initializeColor();
	initializeGrid();
	gameOfLife();

    return 0;
}

// Function to simulate the Game of Life
void gameOfLife()
{
	hipError_t cudaStatus;
	
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for two vectors (one input, one output)
	cudaStatus = hipMalloc((void**)&dev_species, SIZE * numOfSpecies * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_tmp, SIZE * numOfSpecies * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_species, species, SIZE * numOfSpecies * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_tmp, tmp, SIZE * numOfSpecies * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Initialize OpenGL
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
	glutInitWindowSize(WIDTH, HEIGHT);
	glutInitWindowPosition(0, 0);
	glutCreateWindow("Game of Life - Multiple Species");

	// Set timer to recall every 33ms for 30FPS
	glutTimerFunc(1, glutTimer, 1);

	glutKeyboardFunc(keyboard);

	// Set display function that will be called
	glutDisplayFunc(display);

	// Call OpenGL main loop
	glutMainLoop();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}

Error:
	hipFree(dev_species);
	hipFree(dev_tmp);
}

void keyboard(unsigned char key, int x, int y)
{
	switch (key) {
	case(27):
		hipFree(dev_species);
		hipFree(dev_tmp);
		exit(0);
	}
}

void glutTimer(int value)
{
	glutPostRedisplay();
	glutTimerFunc(1, glutTimer, 1);
}

void initializeColor()
{
	for (int i = 0; i < 10; i++)
	{
		color[i] = new float[3];
	}

	color[0][0] = 1.0, color[0][1] = 0.0, color[0][2] = 0.0;
	color[1][0] = 0.0, color[1][1] = 1.0, color[1][2] = 0.0;
	color[2][0] = 0.0, color[2][1] = 0.0, color[2][2] = 1.0;
	color[3][0] = 1.0, color[3][1] = 1.0, color[3][2] = 0.0;
	color[4][0] = 0.0, color[4][1] = 1.0, color[4][2] = 1.0;
	color[5][0] = 1.0, color[5][1] = 0.0, color[5][2] = 1.0;
	color[6][0] = 1.0, color[6][1] = 1.0, color[6][2] = 1.0;
	color[7][0] = 0.5, color[7][1] = 0.75, color[7][2] = 0.33;
	color[8][0] = 0.33, color[8][1] = 0.5, color[8][2] = 0.75;
	color[9][0] = 0.75, color[9][1] = 0.33, color[9][2] = 0.5;
}

void initializeGrid()
{
	species = new bool[numOfSpecies*SIZE];
	tmp = new bool[SIZE*numOfSpecies];

	for (int i = 0; i < numOfSpecies*SIZE; i++)
	{
		species[i] = false;
		tmp[i] = false;
	}

	// Random number generation
	default_random_engine generator;
	uniform_int_distribution<int> distribution(1, numOfSpecies);
	uniform_int_distribution<int> state_distribution(0, 1);

	for (int i = 0; i < SIZE; i++)
	{
		int type = distribution(generator); // Species

		if (state_distribution(generator) == 1)
			species[(type - 1)*SIZE + i] = true; // Alive
										 //else dead
	}
}

void display()
{
	// Call draw function to display grid
	draw();
	glutSwapBuffers();

	//system("pause");

	hipError_t cudaStatus;

	// Call threads for each species to check conditions
	for (int i = 0; i < numOfSpecies; i++)
	{
		countNeighbors<<<1024, 768>>>(dev_species, dev_tmp, i);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		}
	}

	setState<<<2048*numOfSpecies, 384>>>(dev_species, dev_tmp);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	
	cudaStatus = hipMemcpy(species, dev_species, SIZE * numOfSpecies * sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
}

void draw()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glPolygonMode(GL_FRONT_AND_BACK, GL_QUADS);

	// Variables used to draw each pixel and define color
	GLfloat x;
	GLfloat y = 1.0;
	GLfloat red = 0.0;
	GLfloat blue = 0.0;
	GLfloat green = 0.0;
	float factor;

	for (int i = 0; i<HEIGHT; i++)
	{
		x = -1.0;
		for (int j = 0; j<WIDTH; j++)
		{
			factor = 0.0;
			red = 0.0;
			blue = 0.0;
			green = 0.0;

			glBegin(GL_POLYGON);

			//Choose color
			for (int k = 0; k < numOfSpecies; k++)
			{
				if (species[k*SIZE + i*WIDTH + j])
				{
					// Increase the factor based on number of live species on current pixel
					factor++;
					red += color[k][0];
					green += color[k][1];
					blue += color[k][2];
				}
			}
			if (factor != 0)
				glColor3f(red / factor, green / factor, blue / factor);
			else
				glColor3f(red, blue, green); //black
			glVertex2f(x, y - Y_SIZE);
			glVertex2f(x, y);
			glVertex2f(x + X_SIZE, y);
			glVertex2f(x + X_SIZE, y - Y_SIZE);
			glEnd();

			x += X_SIZE;
		}
		y -= Y_SIZE;
	}
}
